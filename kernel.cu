#include <iostream>
#include "kernel.hpp"

void printCudaVersion()
{
    printf("CUDA compiled version %d\n", __CUDACC_VER_MAJOR__);
    
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    int major, minor;
    major = runtime_ver/1000; minor = (runtime_ver%1000)/10;
    printf("CUDA runtime version %d.%d\n", major, minor);

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    major = driver_ver/1000; minor = (driver_ver%1000)/10;
    printf("CUDA driver version %d.%d\n", major, minor);
}